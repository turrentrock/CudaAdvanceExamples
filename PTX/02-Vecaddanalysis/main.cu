#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudaErrors.h>

#define THREAD_MAX 1024

__global__ void Kernel(int* a,int* b,int *c,int n){

	int i = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ extern int shared_mem[];
	int reg;

	if(i>= n) return;

	reg = a[i] + b[i];
	shared_mem[i] = reg;
	c[i] = shared_mem[i];

}

int main(){
	int n=2048;

	int max_threads = THREAD_MAX;
	int max_blocks = ceil(n/max_threads);

	dim3 blocks(max_blocks);
	dim3 threads(max_threads);

	int *a,*b,*c;
	checkCudaErrors(hipMalloc((void **)&a, n*sizeof(*a)));
	checkCudaErrors(hipMalloc((void **)&b, n*sizeof(*b)));
	checkCudaErrors(hipMalloc((void **)&c, n*sizeof(*c)));

	Kernel<<<blocks,threads,n*sizeof(*c)>>>(a,b,c,n);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	checkCudaErrors(hipFree(a));
	checkCudaErrors(hipFree(b));
	checkCudaErrors(hipFree(c));

	return 0;
}
