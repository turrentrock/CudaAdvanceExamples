#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#include <cudaErrors.h>

__global__ void transpose_v0(float* a,float* b, int n){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if(i >= n || j >= n) return;

	b[n*i+j] = a[n*j+i];

}

__global__ void transpose_v1(float* a,float* b, int n){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if(i >= n || j >= n) return;

	b[n*j+i] = a[n*i+j];

}

#define BX 32
#define BY BX

__global__ void transpose_v2(float* a,float* b, int n){

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int i = bx*BX + tx;
	int j = by*BY + ty;

	__shared__ float tile[BY][BX];

	if(i >= n || j >= n) return;

	tile[ty][tx] = a[j*n+i];

	__syncthreads();
	
	i = by*BY + tx;
	j = bx*BX + ty;

	b[j*n+i] = tile[tx][ty];

}

__global__ void transpose_v3(float* a,float* b, int n){

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int i = bx*BX + tx;
	int j = by*BY + ty;

	__shared__ float tile[BY][BX+1]; //Very slight modification to avoid bank conflict in shared mem

	if(i >= n || j >= n) return;

	tile[ty][tx] = a[j*n+i];

	__syncthreads();
	
	i = by*BY + tx;
	j = bx*BX + ty;

	b[j*n+i] = tile[tx][ty];

}


int main(int argc,char** argv){

	int n=16384;
#ifdef UNIT_TEST
	n = 16;
#endif

	int max_thread_per_axis=BX; // max_threads per block is 1024 and sqrt(1024) = 32 
	int max_mem = n*n;

	dim3 blocks(n/max_thread_per_axis+1,n/max_thread_per_axis+1);
	dim3 threads(max_thread_per_axis,max_thread_per_axis);

	float *a;
	checkCudaErrors(hipMallocManaged((void **)&a, max_mem*sizeof(*a)));
	float *b;
	checkCudaErrors(hipMallocManaged((void **)&b, max_mem*sizeof(*b)));

	for(int i=0;i<max_mem;i++) {a[i]=i;b[i]=0;}

	switch(argv[1][0]){
	case '0':
		transpose_v0<<<blocks,threads>>>(a,b,n);
		break;
	case '1':
		transpose_v1<<<blocks,threads>>>(a,b,n);
		break;
	case '2':
		transpose_v2<<<blocks,threads>>>(a,b,n);
		break;
	case '3':
		transpose_v3<<<blocks,threads>>>(a,b,n);
		break;
	}
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

#ifdef UNIT_TEST
	//Expect everything to be n
	std::cout << "----------------------------" << std::endl;
	for(int j=0;j<n;j++){
		for(int i=0;i<n;i++){
			std::cout << b[j*n+i] << " ";
		}
		std::cout<<std::endl;
	}
#endif

	checkCudaErrors(hipFree(a));
	checkCudaErrors(hipFree(b));

	return 0;
}