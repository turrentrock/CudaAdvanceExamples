#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <time.h>

#include <cudaErrors.h> 

#define ASCENDING true
#define DECENDING false

#define THEAD_MAX 1024
#define WARP_SIZE 32

__device__ void exchange(int i,int j,float* arr){
	float temp = arr[i];
	arr[i] = arr[j];
	arr[j] = temp;
}

__device__ void cmp(int i,int j,float* arr,bool direction) {
	if((arr[i] > arr[j]) == direction)
		exchange(i,j,arr);
}

__device__ int power2lessthan(int n){
	int k=1;
	while( k>0 && k < n ){
		k<<=1;
	}

	return k >> 1;
}

#ifdef UNIT_TEST
	#define n 2048
#else
	#define n 67108864 // 2**26 
#endif

__global__ void bitonicSortStep(float* arr,int step){

	int bx = blockIdx.x;
	int tx = threadIdx.x;

	int BX = blockDim.x;

	int i = bx*BX+tx;

	if(i >= n/2) return;

	int direction = (1-(i/step)%2);

	for(int s=step;s>=1;s>>=1){
		__syncthreads();
		int m = i%s;
		int p = (i/s);
		int start = p*s*2 + m;
		cmp(start,start+s,arr,direction);
#ifdef UNIT_TEST
		if(i < n/2)
			printf("%d - %d %d - %f %f - %d\n",i,start,start+s,arr[start],arr[start+s],direction);
#endif

	}
}

int main() {
	srand (time(NULL));

	int max_threads = THEAD_MAX;
	int max_blocks = ceil(n/max_threads);

	dim3 blocks(max_blocks);
	dim3 threads(max_threads);

	float *arr;
	float *d_arr;
	arr = new float[n];

	checkCudaErrors(hipMalloc((void **)&d_arr, n*sizeof(*arr)));

	for(int i=0;i<n;i++) arr[i] = rand() % n;
	checkCudaErrors(hipMemcpy(d_arr,arr,n*sizeof(*arr),hipMemcpyHostToDevice));

#ifdef UNIT_TEST
	//Expect everything to be n
		std::cout << "----------------------------" << std::endl;
		for(int k=0;k<n;k++){
			std::cout << arr[k] << " ";
		}
		std::cout<<std::endl;
#endif

	clock_t start,stop;
	start = clock();

	for(int i=1;i<n;i<<=1){
#ifdef UNIT_TEST
		std::cout << "Step :"<< i << std::endl;
#endif
		bitonicSortStep<<<blocks,threads>>>(d_arr,i);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
#ifdef UNIT_TEST
	//Expect everything to be n
		std::cout << "----------------------------" << std::endl;
		for(int k=0;k<n;k++){
			std::cout << arr[k] << " ";
		}
		std::cout<<std::endl;
#endif

	}
	stop = clock();

	checkCudaErrors(hipMemcpy(arr,d_arr,n*sizeof(*arr),hipMemcpyDeviceToHost));
#ifdef UNIT_TEST
	//Expect everything to be n
		std::cout << "----------------------------" << std::endl;
		for(int k=0;k<n;k++){
			std::cout << arr[k] << " ";
		}
		std::cout<<std::endl;
#endif

	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cout << "Time for gpu_version: "<<timer_seconds << " seconds"  << std::endl;


	checkCudaErrors(hipFree(d_arr));
	delete arr;
	return 0;
}